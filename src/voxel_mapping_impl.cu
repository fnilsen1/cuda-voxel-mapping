#include "hip/hip_runtime.h"
#include "voxel-mapping/voxel_mapping_impl.cuh"

namespace voxel_mapping {

VoxelMappingImpl::VoxelMappingImpl(const VoxelMappingParams& params)
    : resolution_(params.resolution), occupancy_threshold_(params.occupancy_threshold),
        free_threshold_(params.free_threshold)
{
    if (resolution_ <= 0) {
        throw std::invalid_argument("Resolution must be positive");
    }
    CHECK_CUDA_ERROR(hipStreamCreate(&stream_));
    voxel_map_ = std::make_unique<GpuHashMap>(params.chunk_capacity, params.log_odds_occupied, params.log_odds_free, params.log_odds_min, params.log_odds_max, params.occupancy_threshold, params.free_threshold);
    spdlog::info("Voxel map initialized on GPU with initial capacity {}", params.chunk_capacity);
    update_generator_ = std::make_unique<UpdateGenerator>(resolution_, params.min_depth, params.max_depth);
    spdlog::info("Update generator initialized with resolution: {}, min_depth: {}, max_depth: {}", resolution_, params.min_depth, params.max_depth);
    grid_processor_ = std::make_unique<GridProcessor>(params.occupancy_threshold, params.free_threshold, params.edt_max_distance);
    spdlog::info("Grid processor initialized with occupancy threshold: {}, free threshold: {}, EDT max distance: {}", params.occupancy_threshold, params.free_threshold, params.edt_max_distance);
}

VoxelMappingImpl::~VoxelMappingImpl() {
    if (stream_) hipStreamDestroy(stream_);
}

void VoxelMappingImpl::integrate_depth(const float* depth_image, const float* transform) {
    AABBUpdate aabb_update = update_generator_->generate_updates(
        depth_image, 
        transform,
        stream_
    );

    voxel_map_->launch_map_update_kernel(
        aabb_update,
        stream_
    );
    
}

void VoxelMappingImpl::set_camera_properties(float fx, float fy, float cx, float cy, uint32_t width, uint32_t height) {
    update_generator_->set_camera_properties(fx, fy, cx, cy, width, height);
}

AABB VoxelMappingImpl::get_current_aabb() const {
    return update_generator_->get_aabb();
}

Frustum VoxelMappingImpl::get_frustum() const {
    return update_generator_->get_frustum();
}

void VoxelMappingImpl::query_free_chunk_capacity() {
    uint32_t current_freelist_count;
    voxel_map_->get_freelist_counter(&current_freelist_count);
    size_t freelist_capacity = voxel_map_->get_freelist_capacity();
    uint32_t threshold = static_cast<uint32_t>(freelist_capacity * 0.95);

    if (current_freelist_count >= threshold) {
        int3 current_chunk_pos = update_generator_->get_current_chunk_position();
        
        spdlog::info("Freelist usage ({}) is above 95% threshold ({}). Clearing distant chunks.", 
                        current_freelist_count, threshold);

        voxel_map_->clear_chunks(current_chunk_pos);
    }
}

} // namespace voxel_mapping