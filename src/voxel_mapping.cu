#include "hip/hip_runtime.h"
#include "voxel-mapping/voxel_mapping.hpp"
#include <hip/hip_runtime.h>
#include "voxel-mapping/host_macros.hpp"
#include "voxel-mapping/types.hpp"
#include "voxel-mapping/gpu_hash_map.cuh"
#include <vector>
#include <cfloat>
#include <iostream>
#include <spdlog/spdlog.h>
#include "voxel-mapping/update_generator.cuh"
#include "voxel-mapping/grid_processor.cuh"

namespace voxel_mapping {

class VoxelMapping::VoxelMappingImpl {
public:
    float resolution_;
    hipStream_t stream_ = nullptr;
    std::unique_ptr<GpuHashMap> voxel_map_;
    std::unique_ptr<UpdateGenerator> update_generator_;
    std::unique_ptr<GridProcessor> grid_processor_;

    VoxelMappingImpl(size_t map_chunk_capacity, float resolution, float min_depth, float max_depth, VoxelType log_odds_occupied, VoxelType log_odds_free, VoxelType log_odds_min, VoxelType log_odds_max, VoxelType occupancy_threshold, VoxelType free_threshold)
        : resolution_(resolution)
    {
        if (resolution <= 0) {
            throw std::invalid_argument("Resolution must be positive");
        }
        CHECK_CUDA_ERROR(hipStreamCreate(&stream_));
        voxel_map_ = std::make_unique<GpuHashMap>(map_chunk_capacity, log_odds_occupied, log_odds_free, log_odds_min, log_odds_max, occupancy_threshold, free_threshold);
        spdlog::info("Voxel map initialized on GPU with initial capacity {}", map_chunk_capacity);
        update_generator_ = std::make_unique<UpdateGenerator>(resolution_, min_depth, max_depth);
        spdlog::info("Update generator initialized with resolution: {}, min_depth: {}, max_depth: {}", resolution_, min_depth, max_depth);
        grid_processor_ = std::make_unique<GridProcessor>(occupancy_threshold, free_threshold);
        spdlog::info("Grid processor initialized with occupancy threshold: {}, free threshold: {}", occupancy_threshold, free_threshold);
    }

    ~VoxelMappingImpl() {
        if (stream_) hipStreamDestroy(stream_);
    }

    void integrate_depth(const float* depth_image, const float* transform) {
        AABBUpdate aabb_update = update_generator_->generate_updates(
            depth_image, 
            transform,
            stream_
        );

        CHECK_CUDA_ERROR(hipGetLastError()); 

        voxel_map_->launch_map_update_kernel(
            aabb_update,
            stream_
        );
        
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream_));
        CHECK_CUDA_ERROR(hipGetLastError());
    }

    std::vector<VoxelType> extract_grid_block(const AABB& aabb) {
        int aabb_size_x = aabb.size.x;
        int aabb_size_y = aabb.size.y;
        int aabb_size_z = aabb.size.z;
        
        size_t total_elements = aabb_size_x * aabb_size_y * aabb_size_z;

        std::vector<VoxelType> h_block(total_elements);
        VoxelType* d_output_block;
        CHECK_CUDA_ERROR(hipMalloc(&d_output_block, total_elements * sizeof(VoxelType)));

        voxel_map_->extract_block_from_map(d_output_block, aabb);

        CHECK_CUDA_ERROR(hipMemcpy(
            h_block.data(), 
            d_output_block, 
            total_elements * sizeof(VoxelType),
            hipMemcpyDeviceToHost
        ));

        CHECK_CUDA_ERROR(hipFree(d_output_block));
        
        return h_block;
    }

    void extract_esdf_slice(const AABB& aabb_slice, std::vector<int>& esdf_slice) {
        int aabb_size_x = aabb_slice.size.x;
        int aabb_size_y = aabb_slice.size.y;
        int aabb_size_z = aabb_slice.size.z;

        size_t total_elements = aabb_size_x * aabb_size_y * aabb_size_z;
        size_t slice_size = aabb_size_x * aabb_size_y;

        VoxelType* d_aabb;
        CHECK_CUDA_ERROR(hipMalloc(&d_aabb, total_elements * sizeof(VoxelType)));

        voxel_map_->extract_block_from_map(d_aabb, aabb_slice);

        int* d_binary_slice;
        CHECK_CUDA_ERROR(hipMalloc(&d_binary_slice, slice_size * sizeof(int)));

        grid_processor_->launch_extract_binary_slice_kernel(
            d_aabb,
            d_binary_slice,
            aabb_slice.min_corner_index.x,
            aabb_slice.min_corner_index.y,
            aabb_slice.min_corner_index.z,
            aabb_size_x,
            aabb_size_y,
            aabb_size_z,
            stream_
        );

        int* d_esdf_slice;
        CHECK_CUDA_ERROR(hipMalloc(&d_esdf_slice, slice_size * sizeof(int)));

        grid_processor_->launch_edt_kernels(
            d_binary_slice,
            d_esdf_slice,
            aabb_size_x,
            aabb_size_y,
            stream_
        );

        esdf_slice.resize(slice_size);
        CHECK_CUDA_ERROR(hipMemcpy(
            esdf_slice.data(), 
            d_esdf_slice, 
            slice_size * sizeof(int),
            hipMemcpyDeviceToHost
        ));
    }

};

VoxelMapping::VoxelMapping(size_t map_update_capacity, float resolution, float min_depth, float max_depth, VoxelType log_odds_occupied, VoxelType log_odds_free, VoxelType log_odds_min, VoxelType log_odds_max, VoxelType occupancy_threshold, VoxelType free_threshold)
: pimpl_(std::make_unique<VoxelMappingImpl>(map_update_capacity, resolution, min_depth, max_depth, log_odds_occupied, log_odds_free, log_odds_min, log_odds_max, occupancy_threshold, free_threshold))
{
}

VoxelMapping::~VoxelMapping() = default;

VoxelMapping::VoxelMapping(VoxelMapping&&) = default;
VoxelMapping& VoxelMapping::operator=(VoxelMapping&&) = default;

void VoxelMapping::integrate_depth(const float* depth_image, const float* transform) {
    pimpl_->integrate_depth(depth_image, transform);
}

void VoxelMapping::set_camera_properties(float fx, float fy, float cx, float cy, uint32_t width, uint32_t height) {
    pimpl_->update_generator_->set_camera_properties(fx, fy, cx, cy, width, height);
}

std::vector<VoxelType> VoxelMapping::get_3d_block(const AABB& aabb) {
    return pimpl_->extract_grid_block(aabb);
}

void VoxelMapping::extract_esdf_slice(const AABB& aabb, std::vector<int>& esdf_slice) {
    pimpl_->extract_esdf_slice(aabb, esdf_slice);
}

AABB VoxelMapping::get_current_aabb() const {
    AABB aabb;
    int3 min_corner = pimpl_->update_generator_->get_aabb_min_index();
    int3 size = pimpl_->update_generator_->get_aabb_size();
    Vec3i min_corner_index = {min_corner.x, min_corner.y, min_corner.z};
    Vec3i aabb_size = {size.x, size.y, size.z};
    aabb.min_corner_index = min_corner_index;
    aabb.size = aabb_size;
    return aabb;
}

Frustum VoxelMapping::get_frustum() const {
    return pimpl_->update_generator_->get_frustum();
}

} // namespace voxel_mapping